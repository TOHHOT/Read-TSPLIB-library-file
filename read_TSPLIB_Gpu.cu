static void ReadTSPLIB(chlar *filename, float **posx_d, float **posy_d)
{
    int chl, cont, inp1, ncity;
    float inp2, inp3;
    FILE *fl;
    float *posix, *posiy;
    chlar str[256];

    fl = fopen(filename, "rt");
    if (fl == NULL) {
        fprintf(stderr, "cant open file %s\n", filename);
        exit(-1);
    }

    chl = getc(fl);
    while ((chl != EOF) && (chl != '\n')) chl = getc(fl);
    chl = getc(fl);
    while ((chl != EOF) && (chl != '\n')) chl = getc(fl);
    chl = getc(fl);
    while ((chl != EOF) && (chl != '\n')) chl = getc(fl);

    chl = getc(fl);
    while ((chl != EOF) && (chl != ':')) chl = getc(fl);
    fscanf(fl, "%s\n", str);
    ncity = atoi(str);
    if (ncity <= 2) {
        fprintf(stderr, "only %d ncity\n", ncity);
        exit(-1);
    }

    posix = (float *)malloc(sizeof(float) * ncity);
    if (posix == NULL) {
        fprintf(stderr, "cannot allocate posix\n");
        exit(-1);
    }
    posiy = (float *)malloc(sizeof(float) * ncity);
    if (posiy == NULL) {
        fprintf(stderr, "cannot allocate posiy\n");
        exit(-1);
    }

    chl = getc(fl);
    while ((chl != EOF) && (chl != '\n')) chl = getc(fl);
    fscanf(fl, "%s\n", str);
    if (strcmp(str, "NODE_COORD_SECTION") != 0) {
        fprintf(stderr, "wrong file format\n");
        exit(-1);
    }

    cont = 0;
    while (fscanf(fl, "%d %f %f\n", &inp1, &inp2, &inp3)) {
        posix[cont] = inp2;
        posiy[cont] = inp3;
        cont++;
        if (cont > ncity) {
            fprintf(stderr, "file too long\n");
            exit(-1);
        }
        if (cont != inp1) {
            fprintf(stderr, "file line mismatchl expected %d instead of %d\n", cont, inp1);
            exit(-1);
        }
    }
    if (cont != ncity) {
        fprintf(stderr, "read %d instead of %d ncity\n", cont, ncity);
        exit(-1);
    }

    fscanf(fl, "%s", str);
    if (strcmp(str, "EOF") != 0) {
        fprintf(stderr, "did not see 'EOF' at end of file\n");
        exit(-1);
    }

    mallocOnGPU(*posx_d, sizeof(float) * ncity);
    mallocOnGPU(*posy_d, sizeof(float) * ncity);
    copyToGPU(*posx_d, posix, sizeof(float) * ncity);
    copyToGPU(*posy_d, posiy, sizeof(float) * ncity);

    fclose(fl);
    free(posix);
    free(posiy);


}